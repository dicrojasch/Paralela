#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <float.h>


__global__ void matrixMult(const double *A, const double *B, double *C, int N){
	int i, j, firstRow, firstCol, assign, start, totalThreads, id;
	totalThreads = gridDim.x * blockDim.x;
	id = blockDim.x * blockIdx.x  + threadIdx.x;
	if( id < N*N ){
		assign = (N*N) / totalThreads; assign = assign < 1 ? 1:assign;
		start = id * assign;
		for( i = start; i < start+assign; i++ ){
			C[i] = 0; firstRow = (i / N)*N; firstCol = i % N;
			for( j = 0; j < N; j++ )
				C[i] += A[firstRow+j] * B[(j*N)+firstCol];
		}
	}
}

int main1(int argc, char *argv[]){
    hipEvent_t start, stop;
    float elapsedTime;
	hipEventCreate(&start); hipEventRecord(start,0); // Start Measure

	int i, N = atoi(argv[1]), threadsPerBlock = atoi(argv[2]), blocksPerGrid = atoi(argv[3]);
	size_t size = sizeof(double) * (N*N);
	hipError_t err = hipSuccess;
	srand(time(NULL));
	threadsPerBlock = threadsPerBlock*blocksPerGrid > N*N ? (N*N/blocksPerGrid)+1 : threadsPerBlock;

	double *h_A = (double *)malloc(size);
	double *h_B = (double *)malloc(size);
	double *h_C = (double *)malloc(size);
	if (h_A == NULL || h_B == NULL || h_C == NULL){
		fprintf(stderr, "Failed to allocate host Matrix!\n");
		exit(EXIT_FAILURE);
	}

	for( i = 0; i < N*N; i++){
        h_A[i] = rand()/(double)RAND_MAX;
        h_B[i] = rand()/(double)RAND_MAX;
	}

	double *d_A = NULL;
	err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device matrix A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	double *d_B = NULL;
	err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device matrix B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	double *d_C = NULL;
	err = hipMalloc((void **)&d_C, size);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device matrix C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy Matrix A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy Matrix B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	matrixMult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy matrix C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

//    for (int i = 0; i < N*N; i++){
//    	if( i%N == 0)
//    	    printf("\n");
//    	printf("%f ", h_C[i]);
//    }

    err = hipFree(d_A);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_B);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_C);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	free(h_A);
	free(h_B);
	free(h_C);

	hipEventCreate(&stop);  hipEventRecord(stop,0); hipEventSynchronize(stop); // Stop Measure
	hipEventElapsedTime(&elapsedTime, start,stop);
	printf("        Elapsed time : %f ms\n" ,elapsedTime);
	return 0;
}
