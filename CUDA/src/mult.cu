#include "hip/hip_runtime.h"
#include <stdio.h>
#include<sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <float.h>


__global__ void matrixMult(const double *A, const double *B, double *C, int sizeMatrix){
	int i, j;
	int totalThreads = gridDim.x * blockDim.x;
	int id = blockDim.x * blockIdx.x  + threadIdx.x;
	int assign = sizeMatrix / totalThreads;
	assign = assign < 0 ? 1:assign;
	int start = id * assign;
	int firstRow = 0, firstCol = 0;
	for( i = start; i < assign; i++ ){
		C[i] = 0;
		firstRow = (i+1)/ sizeMatrix;
		firstCol = i % sizeMatrix;
		for( j = 0; j < sizeMatrix; j++ )
			C[i] += A[firstRow+j] * B[(j*sizeMatrix)+firstCol];
		if( i > (assign-1) &&  ((assign*totalThreads + id) < sizeMatrix)){
			i = assign*totalThreads + id;
			assign = i + 1;
		}
	}
}

int main(int argc, char *argv[]){
	hipError_t err = hipSuccess;
	int threadsPerBlock = atoi(argv[1]), N = atoi(argv[2]), blocksPerGrid = atoi(argv[3]);
	int i, size;
	srand(time(NULL));

	size = sizeof(double) * (N*N);


	double *h_A = (double *)malloc(size);
	double *h_B = (double *)malloc(size);
	double *h_C = (double *)malloc(size);
	if (h_A == NULL || h_B == NULL || h_C == NULL){
		fprintf(stderr, "Failed to allocate host Matrix!\n");
		exit(EXIT_FAILURE);
	}

	for( i = 0; i < N*N; i++){
		h_A[i] = rand()/(float)RAND_MAX;
		h_B[i] = rand()/(float)RAND_MAX;
	}

	double *d_A = (double *)malloc(size);
	err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device matrix A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	double *d_B = (double *)malloc(size);
	err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device matrix B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	double *d_C = (double *)malloc(size);
	err = hipMalloc((void **)&d_C, size);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device matrix C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_A, d_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy Matrix A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_B, d_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy Matrix B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    matrixMult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N*N);
    err = hipGetLastError();

    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy matrix C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < N; i++){
    	printf("%d ", h_C[i]);
    	if( i%N == 0){
    		printf("\n");
    	}
    }

    err = hipFree(d_A);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_B);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_C);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}
