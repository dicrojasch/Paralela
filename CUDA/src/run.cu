#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


int main1(void){
    hipError_t err = hipSuccess;
    int multiP = 0, cores = 0;

    // Measure time
    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start); hipEventRecord(start,0); // Start Measure

    int deviceCount = 0;
    err = hipGetDeviceCount(&deviceCount);

    if (err != hipSuccess){
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)err, hipGetErrorString(err));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}
    if (deviceCount == 0){
		printf("There are no available device(s) that support CUDA\n");
		exit(1);
	}
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    multiP = deviceProp.multiProcessorCount;
    cores = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
    printf("%d , %d \n",multiP, cores);


    hipEventCreate(&stop);  hipEventRecord(stop,0); hipEventSynchronize(stop); // Stop Measure
    hipEventElapsedTime(&elapsedTime, start,stop);
    printf("Elapsed time : %f ms\n" ,elapsedTime);
    return 0;
}

