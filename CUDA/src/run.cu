#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


int main(void){
    hipError_t err = hipSuccess;
    int N, threads, blocks, multiP = 0, cores = 0, i;
    int deviceCount = 0;
    err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess){
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)err, hipGetErrorString(err));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}
    if (deviceCount == 0){
		printf("There are no available device(s) that support CUDA\n");
		exit(1);
	}
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    multiP = deviceProp.multiProcessorCount;
    cores = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
    printf("Multiprocessors(MP): %d , Cores x MP: %d \n", multiP, cores);
    char commandBase[12] = "./src/mult ";



    for( N = 8; N < 1025; N += N ){
    	printf("Matrix %d x %d\n",N,N);
    	for( blocks = 1; blocks <= multiP; blocks++ ){
    		printf("  Block %d\n",blocks);
    		for( threads = 32; threads < 1025; threads += threads ){
    		    char partialCommand[30] = "";
    		    char toChar[15] = "";
    			printf("    Thread %d\n", threads);
    			strcat( partialCommand, commandBase);
    			sprintf(toChar, "%d %d %d", N, threads, blocks);
    			strcat( partialCommand, toChar);
    			for(i = 1; i < 11; i++ ){
    				printf("      Repeticion %d\n",i);
    				system(partialCommand);
    			}
    		}
    	}
    }



    return 0;
}

